#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test evaluation for software global barrier throughput
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/grid/grid_barrier.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Kernel that iterates through the specified number of software global barriers
 */
__global__
inline
void Kernel(hipLaunchParm lp, GridBarrier global_barrier, int iterations)
{
    for (int i = 0; i < iterations; i++)
    {
        global_barrier.Sync();
    }
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;

    // Defaults
    int iterations = 10000;
    int block_size = 128;
    int grid_size = -1;

    // Initialize command line
    CommandLineArgs args(argc, argv);

    // Get args
    args.GetCmdLineArgument("i", iterations);
    args.GetCmdLineArgument("grid-size", grid_size);
    args.GetCmdLineArgument("block-size", block_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>]"
            "[--i=<iterations>]"
            "[--grid-size<grid-size>]"
            "[--block-size<block-size>]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get device ordinal
    int device_ordinal;
    CubDebugExit(hipGetDevice(&device_ordinal));

    // Get device SM version
    int sm_version;
    CubDebugExit(SmVersion(sm_version, device_ordinal));

    // Get SM properties
    int sm_count, max_block_threads, max_sm_occupancy;
    CubDebugExit(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device_ordinal));
    CubDebugExit(hipDeviceGetAttribute(&max_block_threads, hipDeviceAttributeMaxThreadsPerBlock, device_ordinal));
    CubDebugExit(MaxSmOccupancy(max_sm_occupancy, EmptyKernel<void>, 32));

    // Compute grid size and occupancy
    int occupancy = CUB_MIN((max_block_threads / block_size), max_sm_occupancy);

    if (grid_size == -1)
    {
        grid_size = occupancy * sm_count;
    }
    else
    {
        occupancy = grid_size / sm_count;
    }

    printf("Initializing software global barrier for Kernel<<<%d,%d>>> with %d occupancy\n",
        grid_size, block_size, occupancy);
    fflush(stdout);

    // Init global barrier
    GridBarrierLifetime global_barrier;
    global_barrier.Setup(grid_size);
//
//    // Time kernel
    GpuTimer gpu_timer;
    gpu_timer.Start();
    hipLaunchKernel(HIP_KERNEL_NAME(Kernel),
                    dim3(grid_size),
                    dim3(block_size),
                    0,
                    0,
                    global_barrier,
                    iterations);
    gpu_timer.Stop();

    retval = HipcubDebug(hipDeviceSynchronize());

    // Output timing results
    float avg_elapsed = gpu_timer.ElapsedMillis() / float(iterations);
    printf("%d iterations, %f total elapsed millis, %f avg elapsed millis\n",
        iterations,
        gpu_timer.ElapsedMillis(),
        avg_elapsed);

    return retval;
}
