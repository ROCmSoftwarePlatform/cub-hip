#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/detail/mpl/math.h>

void TestLog2(void)
{
    unsigned int result;
    
    result = thrust::detail::mpl::math::log2<  1>::value;   ASSERT_EQUAL(result, 0);
    result = thrust::detail::mpl::math::log2<  2>::value;   ASSERT_EQUAL(result, 1);
    result = thrust::detail::mpl::math::log2<  3>::value;   ASSERT_EQUAL(result, 1);
    result = thrust::detail::mpl::math::log2<  4>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2<  5>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2<  6>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2<  7>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2<  8>::value;   ASSERT_EQUAL(result, 3);
    result = thrust::detail::mpl::math::log2<  9>::value;   ASSERT_EQUAL(result, 3);
    result = thrust::detail::mpl::math::log2< 15>::value;   ASSERT_EQUAL(result, 3);
    result = thrust::detail::mpl::math::log2< 16>::value;   ASSERT_EQUAL(result, 4);
    result = thrust::detail::mpl::math::log2< 17>::value;   ASSERT_EQUAL(result, 4);
    result = thrust::detail::mpl::math::log2<127>::value;   ASSERT_EQUAL(result, 6);
    result = thrust::detail::mpl::math::log2<128>::value;   ASSERT_EQUAL(result, 7);
    result = thrust::detail::mpl::math::log2<129>::value;   ASSERT_EQUAL(result, 7);
    result = thrust::detail::mpl::math::log2<256>::value;   ASSERT_EQUAL(result, 8);
    result = thrust::detail::mpl::math::log2<511>::value;   ASSERT_EQUAL(result, 8);
    result = thrust::detail::mpl::math::log2<512>::value;   ASSERT_EQUAL(result, 9);
}
DECLARE_UNITTEST(TestLog2);

