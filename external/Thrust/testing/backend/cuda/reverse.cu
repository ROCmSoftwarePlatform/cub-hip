#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/reverse.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator>
__global__
void reverse_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::reverse(exec, first, last);
}


template<typename ExecutionPolicy>
void TestReverseDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int> h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;
  
  thrust::reverse(h_data.begin(), h_data.end());
  hipLaunchKernelGGL(HIP_KERNEL_NAME(reverse_kernel), dim3(1), dim3(1), 0, 0, exec, raw_pointer_cast(d_data.data()), raw_pointer_cast(d_data.data() + d_data.size()));
  
  ASSERT_EQUAL(h_data, d_data);
};


void TestReverseDeviceSeq()
{
  TestReverseDevice(thrust::seq);
}
DECLARE_UNITTEST(TestReverseDeviceSeq);


void TestReverseDeviceDevice()
{
  TestReverseDevice(thrust::device);
}
DECLARE_UNITTEST(TestReverseDeviceDevice);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void reverse_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::reverse_copy(exec, first, last, result);
}


template<typename ExecutionPolicy>
void TestReverseCopyDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::host_vector<int> h_data = unittest::random_integers<int>(n);
  thrust::device_vector<int> d_data = h_data;

  thrust::host_vector<int> h_result(n);
  thrust::device_vector<int> d_result(n);

  thrust::reverse_copy(h_data.begin(), h_data.end(), h_result.begin());
  hipLaunchKernelGGL(HIP_KERNEL_NAME(reverse_copy_kernel), dim3(1), dim3(1), 0, 0, exec, d_data.begin(), d_data.end(), d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
};


void TestReverseCopyDeviceSeq()
{
  TestReverseCopyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestReverseCopyDeviceSeq);


void TestReverseCopyDeviceDevice()
{
  TestReverseCopyDevice(thrust::device);
}
DECLARE_UNITTEST(TestReverseCopyDeviceDevice);


void TestReverseCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  Vector data(5);
  data[0] = 1;
  data[1] = 2;
  data[2] = 3;
  data[3] = 4;
  data[4] = 5;

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::reverse(thrust::cuda::par.on(s), data.begin(), data.end());

  hipStreamSynchronize(s);

  Vector ref(5);
  ref[0] = 5;
  ref[1] = 4;
  ref[2] = 3;
  ref[3] = 2;
  ref[4] = 1;

  ASSERT_EQUAL(ref, data);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestReverseCudaStreams);


void TestReverseCopyCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  Vector data(5);
  data[0] = 1;
  data[1] = 2;
  data[2] = 3;
  data[3] = 4;
  data[4] = 5;

  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::reverse_copy(thrust::cuda::par.on(s), data.begin(), data.end(), result.begin());

  hipStreamSynchronize(s);

  Vector ref(5);
  ref[0] = 5;
  ref[1] = 4;
  ref[2] = 3;
  ref[3] = 2;
  ref[4] = 1;

  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestReverseCopyCudaStreams);

