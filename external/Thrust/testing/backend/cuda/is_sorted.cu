#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename Iterator2>
__global__
void is_sorted_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Iterator2 result)
{
  *result = thrust::is_sorted(exec, first, last);
}


template<typename ExecutionPolicy>
void TestIsSortedDevice(ExecutionPolicy exec)
{
  size_t n = 1000;

  thrust::device_vector<int> v = unittest::random_integers<int>(n);

  thrust::device_vector<bool> result(1);

  v[0] = 1;
  v[1] = 0;

  hipLaunchKernelGGL(HIP_KERNEL_NAME(is_sorted_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin(), v.end(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  thrust::sort(v.begin(), v.end());

  hipLaunchKernelGGL(HIP_KERNEL_NAME(is_sorted_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin(), v.end(), result.begin());
  ASSERT_EQUAL(true, result[0]);
}

void TestIsSortedDeviceSeq()
{
  TestIsSortedDevice(thrust::seq);
}
DECLARE_UNITTEST(TestIsSortedDeviceSeq);


void TestIsSortedDeviceDevice()
{
  TestIsSortedDevice(thrust::device);
}
DECLARE_UNITTEST(TestIsSortedDeviceDevice);


void TestIsSortedCudaStreams()
{
  thrust::device_vector<int> v(4);
  v[0] = 0; v[1] = 5; v[2] = 8; v[3] = 0;

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 0), true);
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 1), true);
  
  // the following line crashes gcc 4.3
#if (__GNUC__ == 4) && (__GNUC_MINOR__ == 3)
  // do nothing
#else
  // compile this line on other compilers
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 2), true);
#endif // GCC

  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 3), true);
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 4), false);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 3, thrust::less<int>()),    true);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 1, thrust::greater<int>()), true);
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.begin() + 4, thrust::greater<int>()), false);
  
  ASSERT_EQUAL(thrust::is_sorted(thrust::cuda::par.on(s), v.begin(), v.end()), false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestIsSortedCudaStreams);

