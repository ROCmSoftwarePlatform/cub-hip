#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/equal.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3>
__global__
void equal_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator3 result)
{
  *result = thrust::equal(exec, first1, last1, first2);
}


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename BinaryPredicate, typename Iterator3>
__global__
void equal_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, BinaryPredicate pred, Iterator3 result)
{
  *result = thrust::equal(exec, first1, last1, first2, pred);
}


template<typename T, typename ExecutionPolicy>
void TestEqualDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::device_vector<T> d_data1 = unittest::random_samples<T>(n);
  thrust::device_vector<T> d_data2 = unittest::random_samples<T>(n);
  thrust::device_vector<bool> d_result(1, false);
  
  //empty ranges
  hipLaunchKernelGGL(HIP_KERNEL_NAME(equal_kernel), dim3(1), dim3(1), 0, 0, exec, d_data1.begin(), d_data1.begin(), d_data1.begin(), d_result.begin());
  ASSERT_EQUAL(d_result[0], true);
  
  //symmetric cases
  hipLaunchKernelGGL(HIP_KERNEL_NAME(equal_kernel), dim3(1), dim3(1), 0, 0, exec, d_data1.begin(), d_data1.end(), d_data1.begin(), d_result.begin());
  ASSERT_EQUAL(d_result[0], true);
  
  if(n > 0)
  {
    d_data1[0] = 0; d_data2[0] = 1;
    
    //different vectors
    hipLaunchKernelGGL(HIP_KERNEL_NAME(equal_kernel), dim3(1), dim3(1), 0, 0, exec, d_data1.begin(), d_data1.end(), d_data2.begin(), d_result.begin());
    ASSERT_EQUAL(d_result[0], false);
    
    //different predicates
    hipLaunchKernelGGL(HIP_KERNEL_NAME(equal_kernel), dim3(1), dim3(1), 0, 0, exec, d_data1.begin(), d_data1.begin() + 1, d_data2.begin(), thrust::less<T>(), d_result.begin());
    ASSERT_EQUAL(d_result[0], true);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(equal_kernel), dim3(1), dim3(1), 0, 0, exec, d_data1.begin(), d_data1.begin() + 1, d_data2.begin(), thrust::greater<T>(), d_result.begin());
    ASSERT_EQUAL(d_result[0], false);
  }
}


template<typename T>
void TestEqualDeviceSeq(const size_t n)
{
  TestEqualDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestEqualDeviceSeq);


template<typename T>
void TestEqualDeviceDevice(const size_t n)
{
  TestEqualDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestEqualDeviceDevice);


void TestEqualCudaStreams()
{
  thrust::device_vector<int> v1(5);
  thrust::device_vector<int> v2(5);
  v1[0] = 5; v1[1] = 2; v1[2] = 0; v1[3] = 0; v1[4] = 0;
  v2[0] = 5; v2[1] = 2; v2[2] = 0; v2[3] = 6; v2[4] = 1;

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v1.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin()), false);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v2.begin(), v2.end(), v2.begin()), true);
  
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 0, v1.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 1, v1.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 3, v2.begin()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.begin() + 4, v2.begin()), false);
  
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin(), thrust::less_equal<int>()), true);
  ASSERT_EQUAL(thrust::equal(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin(), thrust::greater<int>()),    false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestEqualCudaStreams);

