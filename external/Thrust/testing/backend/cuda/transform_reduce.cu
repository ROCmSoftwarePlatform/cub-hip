#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator1, typename Function1, typename T, typename Function2, typename Iterator2>
__global__
void transform_reduce_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Function1 f1, T init, Function2 f2, Iterator2 result)
{
  *result = thrust::transform_reduce(exec, first, last, f1, init, f2);
}


template<typename ExecutionPolicy>
void TestTransformReduceDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  thrust::device_vector<T> result(1);

  hipLaunchKernelGGL(HIP_KERNEL_NAME(transform_reduce_kernel), dim3(1), dim3(1), 0, 0, exec, data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>(), result.begin());
  
  ASSERT_EQUAL(8, (T)result[0]);
}


void TestTransformReduceDeviceSeq()
{
  TestTransformReduceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestTransformReduceDeviceSeq);


void TestTransformReduceDeviceDevice()
{
  TestTransformReduceDevice(thrust::device);
}
DECLARE_UNITTEST(TestTransformReduceDeviceDevice);


void TestTransformReduceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  hipStream_t s;
  hipStreamCreate(&s);

  T result = thrust::transform_reduce(thrust::cuda::par.on(s), data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>());
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(8, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestTransformReduceCudaStreams);

