#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/uninitialized_fill.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void uninitialized_fill_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T val)
{
  thrust::uninitialized_fill(exec, first, last, val);
}


template<typename ExecutionPolicy>
void TestUninitializedFillDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin() + 1, v.begin() + 4, exemplar);
  
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 4);
  
  exemplar = 8;
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin() + 0, v.begin() + 3, exemplar);
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], 7);
  ASSERT_EQUAL(v[4], 4);
  
  exemplar = 9;
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin() + 2, v.end(), exemplar);
  
  ASSERT_EQUAL(v[0], 8);
  ASSERT_EQUAL(v[1], 8);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 9);
  
  exemplar = 1;
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin(), v.end(), exemplar);
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);
}


void TestUninitializedFillDeviceSeq()
{
  TestUninitializedFillDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedFillDeviceSeq);


void TestUninitializedFillDeviceDevice()
{
  TestUninitializedFillDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedFillDeviceDevice);


void TestUninitializedFillCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::uninitialized_fill(thrust::cuda::par.on(s), v.begin(), v.end(), exemplar);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedFillCudaStreams);


template<typename ExecutionPolicy, typename Iterator1, typename Size, typename T, typename Iterator2>
__global__
void uninitialized_fill_n_kernel(ExecutionPolicy exec, Iterator1 first, Size n, T val, Iterator2 result)
{
  *result = thrust::uninitialized_fill_n(exec, first, n, val);
}


template<typename ExecutionPolicy>
void TestUninitializedFillNDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);

  thrust::device_vector<Vector::iterator> iter_vec(1);
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_n_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin() + 1, 3, exemplar, iter_vec.begin());
  Vector::iterator iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 4);
  ASSERT_EQUAL_QUIET(v.begin() + 4, iter);
  
  exemplar = 8;
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_n_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin() + 0, 3, exemplar, iter_vec.begin());
  iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], 7);
  ASSERT_EQUAL(v[4], 4);
  ASSERT_EQUAL_QUIET(v.begin() + 3, iter);
  
  exemplar = 9;
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_n_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin() + 2, 3, exemplar, iter_vec.begin());
  iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], 8);
  ASSERT_EQUAL(v[1], 8);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], 9);
  ASSERT_EQUAL_QUIET(v.end(), iter);
  
  exemplar = 1;
  
  hipLaunchKernelGGL(HIP_KERNEL_NAME(uninitialized_fill_n_kernel), dim3(1), dim3(1), 0, 0, exec, v.begin(), v.size(), exemplar, iter_vec.begin());
  iter = iter_vec[0];
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);
  ASSERT_EQUAL_QUIET(v.end(), iter);
}


void TestUninitializedFillNDeviceSeq()
{
  TestUninitializedFillNDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedFillNDeviceSeq);


void TestUninitializedFillNDeviceDevice()
{
  TestUninitializedFillNDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedFillNDeviceDevice);


void TestUninitializedFillNCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);
  v[0] = 0; v[1] = 1; v[2] = 2; v[3] = 3; v[4] = 4;
  
  T exemplar(7);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::uninitialized_fill_n(thrust::cuda::par.on(s), v.begin(), v.size(), exemplar);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(v[0], exemplar);
  ASSERT_EQUAL(v[1], exemplar);
  ASSERT_EQUAL(v[2], exemplar);
  ASSERT_EQUAL(v[3], exemplar);
  ASSERT_EQUAL(v[4], exemplar);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedFillNCudaStreams);

